
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void calcCenterOfMass(int n, int i, double *theta, double *com_x,
                                 double *com_y, double *R, double *Theta) {
  for (int j = 0; j < n; j++) {
    com_x[i] += cos(theta[j]);
    com_y[i] += sin(theta[j]);
  }
  com_x[i] /= n;
  com_y[i] /= n;
  *R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
  *Theta = atan2(com_y[i], com_x[i]);
}

__global__ void calcThetaDt(int n, double *omega, double *theta, double k,
                            double *R, double *Theta, double *theta_dt) {
  for (int j = 0; j < n; j++) {
    theta_dt[j] = omega[j] + k * (*R) * sin(*Theta - theta[j]);
  }
}

__global__ void calcNextTheta(int n, double *theta, double *theta_dt,
                              double time_delta) {
  for (int j = 0; j < n; j++) {
    theta[j] += theta_dt[j] * time_delta;
  }
}

void simulation(int n, double k, double *omega, double *theta, int loop_count,
                double time_delta, double *com_x, double *com_y,
                double *theta_dt, int verbose) {

  double *d_R;
  double *d_Theta;
  hipError_t error;

  hipMalloc((void **)&d_R, sizeof(double) * 1);
  hipMalloc((void **)&d_Theta, sizeof(double) * 1);

  for (int i = 0; i < loop_count; i++) {
    printf("step: %d\n", i);

    // calc center o fmass
    calcCenterOfMass<<<1, 1>>>(n, i, theta, com_x, com_y, d_R, d_Theta);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != 0) {
      printf("error: %d : %s\n", error, hipGetErrorString(error));
      return;
    }

    // calc theta_dt
    calcThetaDt<<<1, 1>>>(n, omega, theta, k, d_R, d_Theta, theta_dt);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != 0) {
      printf("error: %d : %s\n", error, hipGetErrorString(error));
      return;
    }

    // calc next theta
    calcNextTheta<<<1, 1>>>(n, theta, theta_dt, time_delta);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != 0) {
      printf("error: %d : %s\n", error, hipGetErrorString(error));
      return;
    }
  }
}

double frand() { return (double)rand() / ((double)RAND_MAX + 1); }

double rnorm(double mu, double sigma) {
  return mu + sigma * sqrt(-2.0 * log(frand())) * cos(2.0 * M_PI * frand());
}

void init_variables(const int n, const double mu, const double sigma,
                    const unsigned int seed, double *omega, double *theta) {
  // setup random
  srand(seed);

  for (int i = 0; i < n; i++) {
    omega[i] = rnorm(mu, sigma);
    theta[i] = 2.0 * M_PI * frand();
  }
}

void kuramoto_model_simulator(const int n, const double k,
                              const double time_delta, const int loop_count,
                              const double mu, const double sigma,
                              const unsigned int seed, double *omega,
                              double *theta, double *com_x, double *com_y,
                              int verbose) {

  double *d_omega;
  double *d_theta;
  double *d_com_x;
  double *d_com_y;
  double *d_theta_dt;

  hipMalloc((void **)&d_omega, sizeof(double) * n);
  hipMalloc((void **)&d_theta, sizeof(double) * n);
  hipMalloc((void **)&d_theta_dt, sizeof(double) * n);
  hipMalloc((void **)&d_com_x, sizeof(double) * loop_count);
  hipMalloc((void **)&d_com_y, sizeof(double) * loop_count);
  hipMemset(d_com_x, 0.0, sizeof(double) * loop_count);
  hipMemset(d_com_y, 0.0, sizeof(double) * loop_count);

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  hipMemcpy(d_omega, omega, sizeof(double) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_theta, theta, sizeof(double) * n, hipMemcpyHostToDevice);

  // run simulation
  simulation(n, k, d_omega, d_theta, loop_count, time_delta, d_com_x, d_com_y,
             d_theta_dt, verbose);

  hipMemcpy(com_x, d_com_x, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);
  hipMemcpy(com_y, d_com_y, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  hipFree(d_omega);
  hipFree(d_theta);
  hipFree(d_com_x);
  hipFree(d_com_y);
  hipFree(d_theta_dt);
}

int main(int argc, char const *argv[]) {

  // simulation condition
  const int n = 30;
  const double k = 4;
  const double time_delta = 0.01;
  const int loop_count = 1000;
  const double mu = 1.0;
  const double sigma = 1.0;
  unsigned int seed = (unsigned int)time(NULL);
  int verbose = 0;

  // simulated data
  double *omega;
  double *theta;
  double *com_x;
  double *com_y;

  omega = (double *)calloc(n, sizeof(double));
  theta = (double *)calloc(n, sizeof(double));
  com_x = (double *)calloc(loop_count, sizeof(double));
  com_y = (double *)calloc(loop_count, sizeof(double));

  kuramoto_model_simulator(n, k, time_delta, loop_count, mu, sigma, seed, omega,
                           theta, com_x, com_y, verbose);

  printf("==== output\n");
  for (int i = 0; i < loop_count; i++) {
    double R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    double Theta = atan2(com_y[i], com_x[i]);
    printf("R: %f, Theta: %f, com_x: %f, com_y: %f\n", R, Theta, com_x[i],
           com_y[i]);
  }

  return 0;
}
