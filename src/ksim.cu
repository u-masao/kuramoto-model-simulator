#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void simulation_cu(int n, double k, double *omega, double *theta,
                              int loop_count, double time_delta, int verbose,
                              double *com_x, double *com_y, double *theta_dt,
                              double *R, double *Theta, double *theta_cos,
                              double *theta_sin, int mt_flag) {

  int idx;
  idx = (blockIdx.x * blockDim.x + threadIdx.x) * blockDim.y + threadIdx.y;
  /*
  printf("idx: %d\n", idx);
  printf("bDim.x: %d, bDim.y: %d, bDim.z: %d\n", blockDim.x, blockDim.y,
         blockDim.z);
  printf("bIdx.x: %d, bIdx.y: %d, bIdx.z: %d\n", blockIdx.x, blockIdx.y,
         blockIdx.z);
  printf("tIdx.x: %d, tIdx.y: %d, tIdx.z: %d\n", threadIdx.x, threadIdx.y,
         threadIdx.z);
    */
  if (idx >= n)
    return;

  double sum_theta_cos;
  double sum_theta_sin;
  double s_R;
  double s_Theta;

  if (mt_flag) {
    omega[idx] *= time_delta;
  } else {
    for (int j = 0; j < n; j++) {
      omega[j] *= time_delta;
    }
  }

  for (int i = 0; i < loop_count; i++) {
    // calc center o fmass
    if (mt_flag) {
      theta_cos[idx] = cos(theta[idx]);
      theta_sin[idx] = sin(theta[idx]);
    } else {
      for (int j = 0; j < n; j++) {
        theta_cos[j] = cos(theta[j]);
        theta_sin[j] = sin(theta[j]);
      }
    }

    //__syncthreads();
    //__threadfence();

    sum_theta_cos = 0.0;
    sum_theta_sin = 0.0;
    for (int s = 0; s < n; s++) {
      sum_theta_cos += theta_cos[s];
      sum_theta_sin += theta_sin[s];
    }
    if (mt_flag) {
      /*
printf("step: %d, thread: %d, sum_theta_cos: %f\n", i, idx,
       sum_theta_cos);
       */
    }
    com_x[i] = sum_theta_cos / n;
    com_y[i] = sum_theta_sin / n;

    s_R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    s_Theta = atan2(com_y[i], com_x[i]);

    // calc next theta
    if (mt_flag) {
      theta[idx] +=
          omega[idx] + k * (s_R)*sin(s_Theta - theta[idx]) * time_delta;
    } else {
      for (int j = 0; j < n; j++) {
        theta[j] += omega[j] + k * (s_R)*sin(s_Theta - theta[j]) * time_delta;
      }
    }

    //__syncthreads();
    //__threadfence();
  }
}
void printResultSummary(int tail_count, int loop_count, double *com_x,
                        double *com_y) {
  int start_index;
  start_index = min(tail_count, loop_count);
  printf("==== output\n");
  for (int i = loop_count - start_index; i < loop_count; i++) {
    double R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    double Theta = atan2(com_y[i], com_x[i]);
    printf("R: %f, Theta: %f, com_x: %f, com_y: %f\n", R, Theta, com_x[i],
           com_y[i]);
  }
}

double frand() { return (double)rand() / ((double)RAND_MAX + 1); }

double rnorm(double mu, double sigma) {
  return mu + sigma * sqrt(-2.0 * log(frand())) * cos(2.0 * M_PI * frand());
}

void init_variables(const int n, const double mu, const double sigma,
                    const unsigned int seed, double *omega, double *theta) {
  // setup random
  srand(seed);

  for (int i = 0; i < n; i++) {
    omega[i] = rnorm(mu, sigma);
    theta[i] = 2.0 * M_PI * frand();
  }
}

void kuramoto_model_simulator_cu(const int n, const double k,
                                 const double time_delta, const int loop_count,
                                 const double mu, const double sigma,
                                 const unsigned int seed, double *omega,
                                 double *theta, double *com_x, double *com_y,
                                 int verbose, int mt_flag,
                                 const int blocksize) {

  double *d_omega;
  double *d_theta;
  double *d_com_x;
  double *d_com_y;
  double *d_theta_dt;
  double *d_R;
  double *d_Theta;
  hipError_t error;
  double *d_theta_cos;
  double *d_theta_sin;

  hipMalloc((void **)&d_omega, sizeof(double) * n);
  hipMalloc((void **)&d_theta, sizeof(double) * n);
  hipMalloc((void **)&d_theta_dt, sizeof(double) * n);
  hipMalloc((void **)&d_com_x, sizeof(double) * loop_count);
  hipMalloc((void **)&d_com_y, sizeof(double) * loop_count);
  hipMemset(d_com_x, 0.0, sizeof(double) * loop_count);
  hipMemset(d_com_y, 0.0, sizeof(double) * loop_count);
  hipMalloc((void **)&d_R, sizeof(double) * 1);
  hipMalloc((void **)&d_Theta, sizeof(double) * 1);
  hipMalloc((void **)&d_theta_cos, sizeof(double) * n);
  hipMalloc((void **)&d_theta_sin, sizeof(double) * n);

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  hipMemcpy(d_omega, omega, sizeof(double) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_theta, theta, sizeof(double) * n, hipMemcpyHostToDevice);

  // run simulation
  dim3 block(blocksize, 1, 1);
  dim3 grid(n / blocksize, 1, 1);

  if (mt_flag) {
    simulation_cu<<<grid, block>>>(
        n, k, d_omega, d_theta, loop_count, time_delta, verbose, d_com_x,
        d_com_y, d_theta_dt, d_R, d_Theta, d_theta_cos, d_theta_sin, mt_flag);
  } else {
    simulation_cu<<<1, 1>>>(n, k, d_omega, d_theta, loop_count, time_delta,
                            verbose, d_com_x, d_com_y, d_theta_dt, d_R, d_Theta,
                            d_theta_cos, d_theta_sin, mt_flag);
  }

  error = hipGetLastError();
  if (error != 0) {
    printf("error: %d : %s\n", error, hipGetErrorString(error));
    return;
  }
  hipMemcpy(com_x, d_com_x, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);
  hipMemcpy(com_y, d_com_y, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  hipFree(d_omega);
  hipFree(d_theta);
  hipFree(d_com_x);
  hipFree(d_com_y);
  hipFree(d_theta_dt);
}

void printParams(int n, double *omega, double *theta) {
  for (int i = 0; i < n; i++) {
    printf("omega[%d]: %f, theta[%d]: %f\n", i, omega[i], i, theta[i]);
  }
}

void simulation_c(int n, double k, double *omega, double *theta, int loop_count,
                  double time_delta, double *com_x, double *com_y,
                  int verbose) {
  double R;
  double Theta;
  memset(com_x, 0, loop_count * sizeof(double));
  memset(com_y, 0, loop_count * sizeof(double));

  for (int i = 0; i < loop_count; i++) {
    for (int j = 0; j < n; j++) {
      com_x[i] += cos(theta[j]);
      com_y[i] += sin(theta[j]);
    }
    com_x[i] /= n;
    com_y[i] /= n;
    R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    Theta = atan2(com_y[i], com_x[i]);
    for (int j = 0; j < n; j++) {
      theta[j] += (omega[j] + k * R * sin(Theta - theta[j])) * time_delta;
    }
  }
}

void kuramoto_model_simulator_c(const int n, const double k,
                                const double time_delta, const int loop_count,
                                const double mu, const double sigma,
                                const unsigned int seed, double *omega,
                                double *theta, double *com_x, double *com_y,
                                int verbose, int mt_flag) {

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  // run simulation
  simulation_c(n, k, omega, theta, loop_count, time_delta, com_x, com_y,
               verbose);
}

void printElapsedTime(

    struct timespec *start_time, struct timespec *end_time) {
  unsigned int sec;
  int nsec;
  double d_sec;

  sec = end_time->tv_sec - start_time->tv_sec;
  nsec = end_time->tv_nsec - start_time->tv_nsec;
  d_sec = (double)sec + (double)nsec / (1000 * 1000 * 1000);
  printf("elapsed time: %f\n", d_sec);
}
int main(int argc, char const *argv[]) {

  // simulation condition
  const int blocksize = 1024;
  const int gridsize = 1024;
  const int n = gridsize * blocksize;
  const double k = 4;
  const double time_delta = 0.01;
  const int loop_count = 100;
  const double mu = 1.0;
  const double sigma = 1.0;
  const int verbose = 1;
  const int display_count = 10;
  unsigned int seed = (unsigned int)time(NULL);
  seed = 0;

  struct timespec start_time, end_time;
  int mt_flag = 0;
  // simulated data
  double *omega;
  double *theta;
  double *com_x;
  double *com_y;

  omega = (double *)calloc(n, sizeof(double));
  theta = (double *)calloc(n, sizeof(double));
  com_x = (double *)calloc(loop_count, sizeof(double));
  com_y = (double *)calloc(loop_count, sizeof(double));

  // cpu only
  clock_gettime(CLOCK_REALTIME, &start_time);
  kuramoto_model_simulator_c(n, k, time_delta, loop_count, mu, sigma, seed,
                             omega, theta, com_x, com_y, verbose, mt_flag);
  clock_gettime(CLOCK_REALTIME, &end_time);

  if (verbose > 0) {
    printResultSummary(display_count, loop_count, com_x, com_y);
    printElapsedTime(&start_time, &end_time);
  }

  return 0;

  // single thread
  mt_flag = 0;
  clock_gettime(CLOCK_REALTIME, &start_time);
  kuramoto_model_simulator_cu(n, k, time_delta, loop_count, mu, sigma, seed,
                              omega, theta, com_x, com_y, verbose, mt_flag,
                              blocksize);
  clock_gettime(CLOCK_REALTIME, &end_time);
  if (verbose > 0) {
    printResultSummary(display_count, loop_count, com_x, com_y);
    printElapsedTime(&start_time, &end_time);
  }

  // multi thread
  mt_flag = 1;
  clock_gettime(CLOCK_REALTIME, &start_time);
  kuramoto_model_simulator_cu(n, k, time_delta, loop_count, mu, sigma, seed,
                              omega, theta, com_x, com_y, verbose, mt_flag,
                              blocksize);
  clock_gettime(CLOCK_REALTIME, &end_time);

  if (verbose > 0) {
    printResultSummary(display_count, loop_count, com_x, com_y);
    printElapsedTime(&start_time, &end_time);
  }

  return 0;
}
