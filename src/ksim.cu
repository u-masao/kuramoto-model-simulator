
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void simulation(int n, double k, double *omega, double *theta,
                           int loop_count, double time_delta, double *com_x,
                           double *com_y, double *theta_dt, int verbose) {

  double R;
  double Theta;

  for (int i = 0; i < loop_count; i++) {

    // calc center of mass
    com_x[i] = 0.0;
    com_y[i] = 0.0;
    for (int j = 0; j < n; j++) {
      com_x[i] += cos(theta[j]);
      com_y[i] += sin(theta[j]);
    }
    com_x[i] /= n;
    com_y[i] /= n;

    R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    Theta = atan2(com_y[i], com_x[i]);

    for (int j = 0; j < n; j++) {
      theta_dt[j] = omega[j] + k * R * sin(Theta - theta[j]);
    }
    for (int j = 0; j < n; j++) {
      theta[j] += theta_dt[j] * time_delta;
    }
  }
}

double frand() { return (double)rand() / ((double)RAND_MAX + 1); }

double rnorm(double mu, double sigma) {
  return mu + sigma * sqrt(-2.0 * log(frand())) * cos(2.0 * M_PI * frand());
}

void init_variables(const int n, const double mu, const double sigma,
                    const unsigned int seed, double *omega, double *theta) {
  // setup random
  srand(seed);

  for (int i = 0; i < n; i++) {
    omega[i] = rnorm(mu, sigma);
    theta[i] = 2.0 * M_PI * frand();
  }
}

void kuramoto_model_simulator(const int n, const double k,
                              const double time_delta, const int loop_count,
                              const double mu, const double sigma,
                              const unsigned int seed, double *omega,
                              double *theta, double *com_x, double *com_y,
                              int verbose) {

  double *d_omega;
  double *d_theta;
  double *d_com_x;
  double *d_com_y;
  double *d_theta_dt;

  hipMalloc((void **)&d_omega, sizeof(double) * n);
  hipMalloc((void **)&d_theta, sizeof(double) * n);
  hipMalloc((void **)&d_theta_dt, sizeof(double) * n);
  hipMalloc((void **)&d_com_x, sizeof(double) * loop_count);
  hipMalloc((void **)&d_com_y, sizeof(double) * loop_count);

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  hipMemcpy(d_omega, omega, sizeof(double) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_theta, theta, sizeof(double) * n, hipMemcpyHostToDevice);

  // run simulation
  simulation<<<1, 1>>>(n, k, d_omega, d_theta, loop_count, time_delta, d_com_x,
                       d_com_y, d_theta_dt, verbose);

  hipMemcpy(com_x, d_com_x, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);
  hipMemcpy(com_y, d_com_y, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);

  hipFree(d_omega);
  hipFree(d_theta);
  hipFree(d_com_x);
  hipFree(d_com_y);
  hipFree(d_theta_dt);
}

int main(int argc, char const *argv[]) {
  // simulation condition
  const int n = 30;
  const double k = 4;
  const double time_delta = 0.01;
  const int loop_count = 1000;
  const double mu = 1.0;
  const double sigma = 1.0;
  unsigned int seed = (unsigned int)time(NULL);
  int verbose = 0;

  // simulated data
  double *omega;
  double *theta;
  double *com_x;
  double *com_y;

  omega = (double *)calloc(n, sizeof(double));
  theta = (double *)calloc(n, sizeof(double));
  com_x = (double *)calloc(loop_count, sizeof(double));
  com_y = (double *)calloc(loop_count, sizeof(double));

  kuramoto_model_simulator(n, k, time_delta, loop_count, mu, sigma, seed, omega,
                           theta, com_x, com_y, verbose);

  hipDeviceSynchronize();
  for (int i = 0; i < loop_count; i++) {
    double R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    double Theta = atan2(com_y[i], com_x[i]);
    printf("R: %f, Theta: %f, com_x: %f, com_y: %f\n", R, Theta, com_x[i],
           com_y[i]);
  }

  return 0;
}
