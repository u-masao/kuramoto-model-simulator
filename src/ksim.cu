
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void simulation_cu(int n, double k, double *omega, double *theta,
                              int loop_count, double time_delta, int verbose,
                              double *com_x, double *com_y, double *theta_dt,
                              double *R, double *Theta, double *theta_cos,
                              double *theta_sin, int mt_flag) {

  int j;
  j = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ double sum_theta_cos;
  __shared__ double sum_theta_sin;
  __shared__ double s_R;
  __shared__ double s_Theta;
  if (mt_flag) {
    omega[j] *= time_delta;
  } else {
    for (int j = 0; j < n; j++) {
      omega[j] *= time_delta;
    }
  }

  __syncthreads();
  for (int i = 0; i < loop_count; i++) {
    // calc center o fmass
    if (mt_flag) {
      theta_cos[j] = cos(theta[j]);
      theta_sin[j] = sin(theta[j]);
    } else {
      for (j = 0; j < n; j++) {
        theta_cos[j] = cos(theta[j]);
        theta_sin[j] = sin(theta[j]);
      }
    }
    // printf("step: %d, theta[%d]: %f\n", i, j, theta[j]);
    __syncthreads();

    sum_theta_cos = 0.0;
    sum_theta_sin = 0.0;
    for (int s = 0; s < n; s++) {
      sum_theta_cos += theta_cos[s];
      sum_theta_sin += theta_sin[s];
    }
    com_x[i] = sum_theta_cos / n;
    com_y[i] = sum_theta_sin / n;
    __syncthreads();

    s_R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    s_Theta = atan2(com_y[i], com_x[i]);
    __syncthreads();

    // calc next theta
    if (mt_flag) {
      theta[j] += omega[j] + k * (s_R)*sin(s_Theta - theta[j]) * time_delta;
    } else {
      for (j = 0; j < n; j++) {
        theta[j] += omega[j] + k * (s_R)*sin(s_Theta - theta[j]) * time_delta;
      }
    }

    __syncthreads();
  }
}
void printResultSummary(int tail_count, int loop_count, double *com_x,
                        double *com_y) {
  printf("==== output\n");
  for (int i = loop_count - tail_count; i < loop_count; i++) {
    double R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    double Theta = atan2(com_y[i], com_x[i]);
    printf("R: %f, Theta: %f, com_x: %f, com_y: %f\n", R, Theta, com_x[i],
           com_y[i]);
  }
}

double frand() { return (double)rand() / ((double)RAND_MAX + 1); }

double rnorm(double mu, double sigma) {
  return mu + sigma * sqrt(-2.0 * log(frand())) * cos(2.0 * M_PI * frand());
}

void init_variables(const int n, const double mu, const double sigma,
                    const unsigned int seed, double *omega, double *theta) {
  // setup random
  srand(seed);

  for (int i = 0; i < n; i++) {
    omega[i] = rnorm(mu, sigma);
    theta[i] = 2.0 * M_PI * frand();
  }
}

void kuramoto_model_simulator_cu(const int n, const double k,
                                 const double time_delta, const int loop_count,
                                 const double mu, const double sigma,
                                 const unsigned int seed, double *omega,
                                 double *theta, double *com_x, double *com_y,
                                 int verbose, int mt_flag) {

  double *d_omega;
  double *d_theta;
  double *d_com_x;
  double *d_com_y;
  double *d_theta_dt;
  double *d_R;
  double *d_Theta;
  hipError_t error;
  double *d_theta_cos;
  double *d_theta_sin;

  hipMalloc((void **)&d_omega, sizeof(double) * n);
  hipMalloc((void **)&d_theta, sizeof(double) * n);
  hipMalloc((void **)&d_theta_dt, sizeof(double) * n);
  hipMalloc((void **)&d_com_x, sizeof(double) * loop_count);
  hipMalloc((void **)&d_com_y, sizeof(double) * loop_count);
  hipMemset(d_com_x, 0.0, sizeof(double) * loop_count);
  hipMemset(d_com_y, 0.0, sizeof(double) * loop_count);
  hipMalloc((void **)&d_R, sizeof(double) * 1);
  hipMalloc((void **)&d_Theta, sizeof(double) * 1);
  hipMalloc((void **)&d_theta_cos, sizeof(double) * n);
  hipMalloc((void **)&d_theta_sin, sizeof(double) * n);

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  hipMemcpy(d_omega, omega, sizeof(double) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_theta, theta, sizeof(double) * n, hipMemcpyHostToDevice);

  // run simulation
  int blocksize = 4;
  dim3 block(blocksize, 1, 1);
  dim3 grid(n / block.x, 1, 1);

  simulation_cu<<<grid, block>>>(n, k, d_omega, d_theta, loop_count, time_delta,
                                 verbose, d_com_x, d_com_y, d_theta_dt, d_R,
                                 d_Theta, d_theta_cos, d_theta_sin, mt_flag);

  error = hipGetLastError();
  if (error != 0) {
    printf("error: %d : %s\n", error, hipGetErrorString(error));
    return;
  }
  hipMemcpy(com_x, d_com_x, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);
  hipMemcpy(com_y, d_com_y, sizeof(double) * loop_count,
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  hipFree(d_omega);
  hipFree(d_theta);
  hipFree(d_com_x);
  hipFree(d_com_y);
  hipFree(d_theta_dt);
}

void printParams(int n, double *omega, double *theta) {
  for (int i = 0; i < n; i++) {
    printf("omega[%d]: %f, theta[%d]: %f\n", i, omega[i], i, theta[i]);
  }
}

void calcCenterOfMass(double *theta, int n, double *com_x, double *com_y) {
  *com_x = 0.0;
  *com_y = 0.0;
  for (int i = 0; i < n; i++) {
    *com_x += cos(theta[i]);
    *com_y += sin(theta[i]);
  }
  *com_x /= n;
  *com_y /= n;
}

void kuramoto_formula_fast(int n, double k, double *omega, double *theta,
                           double R, double Theta, double *theta_dt) {
  for (int i = 0; i < n; i++) {
    theta_dt[i] = omega[i] + k * R * sin(Theta - theta[i]);
  }
}

void simulation_c(int n, double k, double *omega, double *theta, int loop_count,
                  double time_delta, double *com_x, double *com_y,
                  int verbose) {
  double R;
  double Theta;
  double *theta_dt;
  theta_dt = (double *)malloc(n * sizeof(double));

  for (int i = 0; i < loop_count; i++) {
    calcCenterOfMass(theta, n, &com_x[i], &com_y[i]);
    R = sqrt(pow(com_x[i], 2) + pow(com_y[i], 2));
    Theta = atan2(com_y[i], com_x[i]);
    kuramoto_formula_fast(n, k, omega, theta, R, Theta, theta_dt);
    for (int j = 0; j < n; j++) {
      theta[j] += theta_dt[j] * time_delta;
    }
  }

  free(theta_dt);
}

void kuramoto_model_simulator_c(const int n, const double k,
                                const double time_delta, const int loop_count,
                                const double mu, const double sigma,
                                const unsigned int seed, double *omega,
                                double *theta, double *com_x, double *com_y,
                                int verbose, int mt_flag) {

  // init variables
  init_variables(n, mu, sigma, seed, omega, theta);

  // run simulation
  simulation_c(n, k, omega, theta, loop_count, time_delta, com_x, com_y,
               verbose);
}

int main(int argc, char const *argv[]) {

  // simulation condition
  const int n = 2 * 4;
  const double k = 4;
  const double time_delta = 0.01;
  const int loop_count = 1000;
  const double mu = 1.0;
  const double sigma = 1.0;
  unsigned int seed = (unsigned int)time(NULL);
  int verbose = 1;
  int mt_flag = 0;

  // simulated data
  double *omega;
  double *theta;
  double *com_x;
  double *com_y;

  omega = (double *)calloc(n, sizeof(double));
  theta = (double *)calloc(n, sizeof(double));
  com_x = (double *)calloc(loop_count, sizeof(double));
  com_y = (double *)calloc(loop_count, sizeof(double));

  kuramoto_model_simulator_c(n, k, time_delta, loop_count, mu, sigma, seed,
                             omega, theta, com_x, com_y, verbose, mt_flag);

  if (verbose > 0) {
    printResultSummary(5, loop_count, com_x, com_y);
  }

  mt_flag = 0;
  kuramoto_model_simulator_cu(n, k, time_delta, loop_count, mu, sigma, seed,
                              omega, theta, com_x, com_y, verbose, mt_flag);
  if (verbose > 0) {
    printResultSummary(5, loop_count, com_x, com_y);
  }
  mt_flag = 1;
  kuramoto_model_simulator_cu(n, k, time_delta, loop_count, mu, sigma, seed,
                              omega, theta, com_x, com_y, verbose, mt_flag);
  if (verbose > 0) {
    printResultSummary(5, loop_count, com_x, com_y);
  }

  return 0;
}
